#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
//#include "d_colorToGreyscale.h"
#include "CHECK.h"

#define CHANNELS 3
#define MASKSIZE 5
#define SUBSAMPLESIZE 2
#define TILEWIDTH 16

__global__ void d_convLayerForwardKernel(int, int, int, unsigned char *, float *, float *);

/**
 * Performs one forward run through the network
 * @param Pin input image
 * @param resulting vector
 * @param size of input image
 */
void d_convLayerForward(unsigned char * inputMap, float * outputMap, float * weights, 
                                            int numInput, float * result)
{
    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing
    //create event objects
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));

    //Create device vectors 
    unsigned char * d_inputMap;
    float * d_weights;
    float * d_outputMap;
    
    int inSize = sizeof(unsigned char)*numInput*numInput;
    int outSize = sizeof(float)*(numInput-MASKSIZE-1)*(numInput-MASKSIZE-1);
    int weightSize = sizeof(float)*25;
    CHECK(hipMalloc((void **)&d_outputMap, outSize));
    CHECK(hipMalloc((void **)&d_weights, weightSize));
    CHECK(hipMalloc((void **)&d_inputMap, inSize));

    CHECK(hipMemcpy(d_inputMap, inputMap, inSize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_weights, weights, weightSize, hipMemcpyHostToDevice));

    //Prepare Convolution Kernel
    outSize = numInput - (MASKSIZE-1);
    int gridSize = outSize/TILEWIDTH;
    int gridZ = gridSize * gridSize;
    dim3 blockDim(TILEWIDTH, TILEWIDTH, 1);
    dim3 gridDim(gridSize, gridSize, gridZ);
    size_t shmemSize = sizeof(float) * ((TILEWIDTH + gridSize-1)*(TILEWIDTH + gridSize-1) + gridSize*gridSize);
    //Launch
    d_convLayerForwardKernel<<<gridDim, blockDim, shmemSize>>>(gridSize, numInput, gridSize, d_inputMap, 
										d_weights, d_outputMap);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(outputMap, d_outputMap, outSize, hipMemcpyDeviceToHost));
    CHECK(hipFree(d_outputMap));
    CHECK(hipFree(d_inputMap));
    CHECK(hipFree(d_weights));
}



/**
 * Convolutes a set of input feature maps into a set
 * of output feature maps
 * @param W_grid width of grid
 * @param numOutput number of output elements
 * @param numInput number of input elements
 * @param inputMap input feature maps
 * @param weights to apply to each input map
 * @param outputMap
 */
__global__ void d_convLayerForwardKernel(int gridWidth, int numInput, int numOutput, unsigned char * inputMap, 
                                                          float * weights, float * outputMap)
{
    int n, m, h_base, w_base, h, w;
    int xTileWidth = TILEWIDTH + numOutput-1;
    int weightLen = xTileWidth * xTileWidth;
    int inputLen = numInput*numInput; 
    extern __shared__ float shmem[];
    float * inputShared = &shmem[0];
    float * weightShared = &shmem[xTileWidth * xTileWidth];    

    n = blockIdx.x;
    m = blockIdx.y;
    h_base = (blockIdx.z / gridWidth) * TILEWIDTH; //vertical base out data index for the block
    w_base = (blockIdx.z % gridWidth) * TILEWIDTH; // horizontal base out data index for the block
    h = h_base + threadIdx.x;
    w = w_base + threadIdx.y;

    float acc = 0.;
    int c, i, j, p, q;
    //Add over all channels
    for (c = 0; c < numInput; c++) {
        //Load weight vector into shared memory
        if ((threadIdx.x < numOutput) && (threadIdx.y < numOutput))                           
            weightShared[threadIdx.y*blockDim.x+threadIdx.x] = 
                           weights[m+numOutput*(c+numInput*(threadIdx.x+blockDim.x*threadIdx.y))]; //m,c,tIdx,tIdy
        __syncthreads();                    
        
        //Load input map into shared memory
        for (i = h; i < h_base + xTileWidth; i += TILEWIDTH) {
            for (j = w; j < w_base + xTileWidth; j += TILEWIDTH)
                inputShared[(i-h_base)*inputLen+(j-w_base)] = (float) inputMap[n+numOutput*(c+CHANNELS*(h+gridWidth*w))]; //n,c,h,w
        }                                       

        __syncthreads();
        for (p = 0; p < numOutput; p++) {
            for (q = 0; q < numOutput; q++)
                acc += inputShared[(h+p)*inputLen+(w+q)] * weightShared[p*weightLen+q];
        }
        __syncthreads();
    }
    outputMap[n+numOutput*(m+numOutput*(h+gridWidth*w))] = acc; //n,m,h,w
}
